
#include <hip/hip_runtime.h>
// compile with: nvcc growcut_3D_euclid_4_channels_constant_penalty.cu -arch sm_20 -ptx 

// info on thread indices:  http://stackoverflow.com/questions/7318002/3d-image-indices

__global__ void update_old_step ( const float          *d_new_strength,
                                       float			   *d_strength,
                                  const unsigned char     *d_new_label,							
                                        unsigned char		  *d_label,

                                  const int                      xDim, 
                                  const int                      yDim,
                                  const int                      zDim)

{	
	  
	  unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	  unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	  unsigned int z = blockDim.z * blockIdx.z + threadIdx.z;

	  if ( (x >= xDim) || (y >= yDim) || (z >= zDim) ) return;
    
	  unsigned int offset = x + (xDim * y) + (xDim *yDim * z);

      //update old step
	  d_strength[offset] = d_new_strength[offset];	
	  d_label[offset]    = d_new_label[offset];

}




__global__ void evol_cells(  const float  			  		    *d_channelOne,
							 const float  			  		    *d_channelTwo,
							 const float                      *d_channelThree,
							 const float                       *d_channelFour,
                             
                             const float 					      *d_strength,
						     const unsigned char 			         *d_label,
                             	    
                             	   float 			  	      *d_new_strength,
							 	   unsigned char 	  	         *d_new_label,
							       unsigned char 	        *d_still_updating,

							 const float                        *norm_2_means,
							 const float                        *norm_2_sdevs,
							 
							 const float 								ndevs,
 							 const float 						 high_penalty,
 							 const float 						  low_penalty,

                             const int                                   xDim, 
							 const int                                   yDim, 
							 const int                                   zDim,
							 
							 const float                            max_norm_2)
{

  
  unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int z = blockDim.z * blockIdx.z + threadIdx.z;

  if ( (x >= xDim) || (y >= yDim) || (z >= zDim) ) 
  	return;
      
   unsigned int offset = x + (xDim * y) + (xDim * yDim * z);
  
   //3D Von Neumann neighborhood
   int left   = (x - 1) + (xDim * y) + (xDim * yDim * z);
   int right  = (x + 1) + (xDim * y) + (xDim * yDim * z);
   int top    = x + (xDim * (y - 1)) + (xDim * yDim * z);
   int bottom = x + (xDim * (y + 1)) + (xDim * yDim * z);
   int front  = x + (xDim * y) + (xDim * yDim * (z - 1));
   int back   = x + (xDim * y) + (xDim * yDim * (z + 1));


   int neigh          = -1;
   int neigh_label    = -1;
   float gfunc        = -1.0f;
   float eval_product = -1.0f;
   
   //will change to 1 if the cell is overpowered by an attacker
   d_still_updating[offset] = 0;
   
   float two = 2.0;

   float distance_to_mean = 0.0f;

   float x_distance       = 0.0f;
   
   float norm_2_neigh     = 0.0f;

   //check left neighbor
   if (x != 0){
	   
	   neigh = left;
      
	   /*begin eval region*/
	   neigh_label = d_label[neigh];
	   	   
	   x_distance = sqrt ( 

							 pow( (d_channelOne[offset]   - d_channelOne[neigh])  , two ) 
															+
							 pow( (d_channelTwo[offset]   - d_channelTwo[neigh])  , two )
															+
							 pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
															+
							 pow( (d_channelFour[offset]  - d_channelFour[neigh]) , two )
	   					
	   	                  );

	   gfunc = 1.0f - ( x_distance / max_norm_2);

	   norm_2_neigh = sqrt (
	   
							pow( (d_channelOne[offset])  , two ) 
											+
	   	                    pow( (d_channelTwo[offset])  , two )
											+
	   	                    pow( (d_channelThree[offset]), two )
											+
	   	                    pow( (d_channelFour[offset]) , two )
	   
							);
	   
	   distance_to_mean = abs (norm_2_neigh - norm_2_means[neigh_label]);

	   //high penalty to gfunc if the neighbor's distance to its mean is beyond (n + 1) standard deviations
	   if ( distance_to_mean > ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) ) {
	   		
	   			gfunc = gfunc * high_penalty;

	   }

       //low penalty to gfunc if the neighbor's distance to its mean is beyond n standard deviations, but less than n + 1
	   if ( 
	   	  	( distance_to_mean > ( ndevs * norm_2_sdevs[neigh_label])     )  
	   									&& 
	      	( distance_to_mean < ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) )
	   ) {
	   		
	   			gfunc = gfunc * low_penalty;

	   }

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered by the strongest neighbor so far
	   if( ( eval_product > d_strength[offset]) && ( eval_product > d_new_strength[offset] ) ) {
	   
		   d_new_strength[offset]   = eval_product;
		   d_new_label[offset]      = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
   }

   //check right neighbor
   if (x != (xDim - 1)){
		
		neigh = right;
		
	   /*begin eval region*/
	   neigh_label = d_label[neigh];
	   	   
	   x_distance = sqrt ( 

							 pow( (d_channelOne[offset]   - d_channelOne[neigh])  , two ) 
															+
							 pow( (d_channelTwo[offset]   - d_channelTwo[neigh])  , two )
															+
							 pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
															+
							 pow( (d_channelFour[offset]  - d_channelFour[neigh]) , two )
	   					
	   	                  );

	   gfunc = 1.0f - ( x_distance / max_norm_2);

	   norm_2_neigh = sqrt (
	   
							pow( (d_channelOne[offset])  , two ) 
											+
	   	                    pow( (d_channelTwo[offset])  , two )
											+
	   	                    pow( (d_channelThree[offset]), two )
											+
	   	                    pow( (d_channelFour[offset]) , two )
	   
							);
	   
	   distance_to_mean = abs (norm_2_neigh - norm_2_means[neigh_label]);

	   //high penalty to gfunc if the neighbor's distance to its mean is beyond (n + 1) standard deviations
	   if ( distance_to_mean > ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) ) {
	   		
	   			gfunc = gfunc * high_penalty;

	   }

       //low penalty to gfunc if the neighbor's distance to its mean is beyond n standard deviations, but less than n + 1
	   if ( 
	   	  	( distance_to_mean > ( ndevs * norm_2_sdevs[neigh_label])       )  
	   									&& 
	      	( distance_to_mean < ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) )
	   ) {
	   		
	   			gfunc = gfunc * low_penalty;

	   }

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered by the strongest neighbor so far
	   if( ( eval_product > d_strength[offset]) && ( eval_product > d_new_strength[offset] ) ) {
	   
		   d_new_strength[offset]   = eval_product;
		   d_new_label[offset]      = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
   }

   //check top neighbor
   if(y != 0){
	   
	   neigh = top;

	   /*begin eval region*/
	   neigh_label = d_label[neigh];
	   	   
	   x_distance = sqrt ( 

							 pow( (d_channelOne[offset]   - d_channelOne[neigh])  , two ) 
															+
							 pow( (d_channelTwo[offset]   - d_channelTwo[neigh])  , two )
															+
							 pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
															+
							 pow( (d_channelFour[offset]  - d_channelFour[neigh]) , two )
	   					
	   	                  );

	   gfunc = 1.0f - ( x_distance / max_norm_2);

	   norm_2_neigh = sqrt (
	   
							pow( (d_channelOne[offset])  , two ) 
											+
	   	                    pow( (d_channelTwo[offset])  , two )
											+
	   	                    pow( (d_channelThree[offset]), two )
											+
	   	                    pow( (d_channelFour[offset]) , two )
	   
							);
	   
	   distance_to_mean = abs (norm_2_neigh - norm_2_means[neigh_label]);

	   //high penalty to gfunc if the neighbor's distance to its mean is beyond (n + 1) standard deviations
	   if ( distance_to_mean > ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) ) {
	   		
	   			gfunc = gfunc * high_penalty;

	   }

       //low penalty to gfunc if the neighbor's distance to its mean is beyond n standard deviations, but less than n + 1
	   if ( 
	   	  	( distance_to_mean > ( ndevs * norm_2_sdevs[neigh_label])     )  
	   									&& 
	      	( distance_to_mean < ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) )
	   ) {
	   		
	   			gfunc = gfunc * low_penalty;

	   }

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered by the strongest neighbor so far
	   if( ( eval_product > d_strength[offset]) && ( eval_product > d_new_strength[offset] ) ) {
	   
		   d_new_strength[offset]   = eval_product;
		   d_new_label[offset]      = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
		
   }

   //check bottom neighbor
   if (y != (yDim - 1)){
		
		neigh = bottom;
		
	   /*begin eval region*/
	   neigh_label = d_label[neigh];
	   	   
	   x_distance = sqrt ( 

							 pow( (d_channelOne[offset]   - d_channelOne[neigh])  , two ) 
															+
							 pow( (d_channelTwo[offset]   - d_channelTwo[neigh])  , two )
															+
							 pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
															+
							 pow( (d_channelFour[offset]  - d_channelFour[neigh]) , two )
	   					
	   	                  );

	   gfunc = 1.0f - ( x_distance / max_norm_2);

	   norm_2_neigh = sqrt (
	   
							pow( (d_channelOne[offset])  , two ) 
											+
	   	                    pow( (d_channelTwo[offset])  , two )
											+
	   	                    pow( (d_channelThree[offset]), two )
											+
	   	                    pow( (d_channelFour[offset]) , two )
	   
							);
	   
	   distance_to_mean = abs (norm_2_neigh - norm_2_means[neigh_label]);

	   //high penalty to gfunc if the neighbor's distance to its mean is beyond (n + 1) standard deviations
	   if ( distance_to_mean > ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) ) {
	   		
	   			gfunc = gfunc * high_penalty;

	   }

       //low penalty to gfunc if the neighbor's distance to its mean is beyond n standard deviations, but less than n + 1
	   if ( 
	   	  	( distance_to_mean > ( ndevs * norm_2_sdevs[neigh_label])     )  
	   									&& 
	      	( distance_to_mean < ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) )
	   ) {
	   		
	   			gfunc = gfunc * low_penalty;

	   }

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered by the strongest neighbor so far
	   if( ( eval_product > d_strength[offset]) && ( eval_product > d_new_strength[offset] ) ) {
	   
		   d_new_strength[offset]   = eval_product;
		   d_new_label[offset]      = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
   }

   //check front neighbor
   if(z != 0){
		
		neigh = front;

	   /*begin eval region*/
	   neigh_label = d_label[neigh];
	   	   
	   x_distance = sqrt ( 

							 pow( (d_channelOne[offset]   - d_channelOne[neigh])  , two ) 
															+
							 pow( (d_channelTwo[offset]   - d_channelTwo[neigh])  , two )
															+
							 pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
															+
							 pow( (d_channelFour[offset]  - d_channelFour[neigh]) , two )
	   					
	   	                  );

	   gfunc = 1.0f - ( x_distance / max_norm_2);

	   norm_2_neigh = sqrt (
	   
							pow( (d_channelOne[offset])  , two ) 
											+
	   	                    pow( (d_channelTwo[offset])  , two )
											+
	   	                    pow( (d_channelThree[offset]), two )
											+
	   	                    pow( (d_channelFour[offset]) , two )
	   
							);
	   
	   distance_to_mean = abs (norm_2_neigh - norm_2_means[neigh_label]);

	   //high penalty to gfunc if the neighbor's distance to its mean is beyond (n + 1) standard deviations
	   if ( distance_to_mean > ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) ){
	   		
	   			gfunc = gfunc * high_penalty;

	   }

       //low penalty to gfunc if the neighbor's distance to its mean is beyond n standard deviations, but less than n + 1
	   if ( 
	   	  	( distance_to_mean > ( ndevs * norm_2_sdevs[neigh_label])     )  
	   									&& 
	      	( distance_to_mean < ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) )
	   ) {
	   		
	   			gfunc = gfunc * low_penalty;

	   }

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered by the strongest neighbor so far
	   if( ( eval_product > d_strength[offset]) && ( eval_product > d_new_strength[offset] ) ) {
	   
		   d_new_strength[offset]   = eval_product;
		   d_new_label[offset]      = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
		
   }

   //check back neighbor
   if(z != (zDim - 1)){
		
		neigh = back;

	   /*begin eval region*/
	   neigh_label = d_label[neigh];
	   	   
	   x_distance = sqrt ( 

							 pow( (d_channelOne[offset]   - d_channelOne[neigh])  , two ) 
															+
							 pow( (d_channelTwo[offset]   - d_channelTwo[neigh])  , two )
															+
							 pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
															+
							 pow( (d_channelFour[offset]  - d_channelFour[neigh]) , two )
	   					
	   	                  );

	   gfunc = 1.0f - ( x_distance / max_norm_2);

	   norm_2_neigh = sqrt (
	   
							pow( (d_channelOne[offset])  , two ) 
											+
	   	                    pow( (d_channelTwo[offset])  , two )
											+
	   	                    pow( (d_channelThree[offset]), two )
											+
	   	                    pow( (d_channelFour[offset]) , two )
	   
							);
	   
	   distance_to_mean = abs (norm_2_neigh - norm_2_means[neigh_label]);

	   //high penalty to gfunc if the neighbor's distance to its mean is beyond (n + 1) standard deviations
	   if ( distance_to_mean > ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) ){
	   		
	   			gfunc = gfunc * high_penalty;

	   }

       //low penalty to gfunc if the neighbor's distance to its mean is beyond n standard deviations, but less than n + 1
	   if ( 
	   	  	( distance_to_mean > ( ndevs * norm_2_sdevs[neigh_label])     )  
	   									&& 
	      	( distance_to_mean < ( (ndevs + 1.0) * norm_2_sdevs[neigh_label]) )
	   ) {
	   		
	   			gfunc = gfunc * low_penalty;

	   }

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered by the strongest neighbor so far
	   if( ( eval_product > d_strength[offset]) && ( eval_product > d_new_strength[offset] ) ) {
	   
		   d_new_strength[offset]   = eval_product;
		   d_new_label[offset]      = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
		
 }
}


	
