
#include <hip/hip_runtime.h>
// compile with: nvcc growcut_3D_euclid_4_channels_vezhnevets.cu -arch sm_20 -ptx 
// info on thread indices: http://stackoverflow.com/questions/7318002/3d-image-indices

__global__ void update_old_step ( const float          *d_new_strength,
                                  float				       *d_strength,
                                  const unsigned char     *d_new_label,							
                                  unsigned char				  *d_label,

                                  const int                      xDim, 
                                  const int                      yDim,
                                  const int                      zDim)

{	
	  //important: x = col, y = row
	  //xDim = col_dim, yDim = row_dim
	  unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	  unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	  unsigned int z = blockDim.z * blockIdx.z + threadIdx.z;

	  if ( (x >= xDim) || (y >= yDim) || (z >= zDim) ) return;
    
	  unsigned int offset = x + (xDim * y) + (xDim *yDim * z);

	  //if the thread id expressed in offset is out of bounds, abort thread execution
	   int nnodes = xDim * yDim * zDim;
	   
	   if (offset >= nnodes) return; 

       //update old step
	   d_strength[offset] = d_new_strength[offset];	
	   d_label[offset] = d_new_label[offset];

}


__global__ void evol_cells(  const float  			  		   *d_channelOne,
							 const float  			  		   *d_channelTwo,
							 const float                     *d_channelThree,
							 const float                      *d_channelFour,
                             
                             const float 					     *d_strength,
						     const unsigned char 			        *d_label,
                             	   
                             	   float 				     *d_new_strength,
							 	   unsigned char 		        *d_new_label,
							       unsigned char 	       *d_still_updating,
							
                             const int xDim, 
							 const int yDim, 
							 const int zDim,
							 const float max_I)
{	  
  
  unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int z = blockDim.z * blockIdx.z + threadIdx.z;

  if ( (x >= xDim) || (y >= yDim) || (z >= zDim) )
  {return;}
    
  unsigned int offset = x + (xDim * y) + (xDim * yDim * z);

   //3D Von Neumann neighborhood
   unsigned int left   = (x - 1) + (xDim * y) + (xDim * yDim * z);
   unsigned int right  = (x + 1) + (xDim * y) + (xDim * yDim * z);
   unsigned int top    = x + (xDim * (y - 1)) + (xDim * yDim * z);
   unsigned int bottom = x + (xDim * (y + 1)) + (xDim * yDim * z);
   unsigned int front  = x + (xDim * y) + (xDim * yDim * (z - 1));
   unsigned int back   = x + (xDim * y) + (xDim * yDim * (z + 1));


   int neigh          = -1;
   
   float gfunc        = -1.0f;
   float eval_product = -1.0f;
   
   //will change to 1 if the cell is overpowered by an attacker
   d_still_updating[offset] = 0;
   
   float two = 2.0;
   //check left neighbor
   if (x != 0){
	   
	   neigh = left;
      
	   /*begin eval region*/
	   gfunc = 1.0f - ( sqrt ( 

	   					pow( (d_channelOne[offset] - d_channelOne[neigh]), two ) 
	   											   +
	   	                pow( (d_channelTwo[offset] - d_channelTwo[neigh]), two )
	   	                						   +
	   	                pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
	   	                						   +
	   	                pow( (d_channelFour[offset] - d_channelFour[neigh]), two )
	   					
	   					)

	               / max_I);

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered 
	   if( ( eval_product > d_strength[offset])){
	   
		   d_new_strength[offset] = eval_product;
		   d_new_label[offset] = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
   }

   //check right neighbor
   if (x != (xDim - 1)){
		
		neigh = right;
		
	   /*begin eval region*/
	   gfunc = 1.0f - ( sqrt ( 

	   					pow( (d_channelOne[offset] - d_channelOne[neigh]), two ) 
	   											   +
	   	                pow( (d_channelTwo[offset] - d_channelTwo[neigh]), two )
	   	                						   +
	   	                pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
	   	                						   +
	   	                pow( (d_channelFour[offset] - d_channelFour[neigh]), two )
	   					
	   					)

	               / max_I);

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered 
	   if( ( eval_product > d_strength[offset])){
	   
		   d_new_strength[offset] = eval_product;
		   d_new_label[offset] = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
   }

   //check top neighbor
   if(y != 0){
	   
	   neigh = top;

	   /*begin eval region*/
	   gfunc = 1.0f - ( sqrt ( 

	   					pow( (d_channelOne[offset] - d_channelOne[neigh]), two ) 
	   											   +
	   	                pow( (d_channelTwo[offset] - d_channelTwo[neigh]), two )
	   	                						   +
	   	                pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
	   	                						   +
	   	                pow( (d_channelFour[offset] - d_channelFour[neigh]), two )
	   					
	   					)

	               / max_I);

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered 
	   if( ( eval_product > d_strength[offset])){
	   
		   d_new_strength[offset] = eval_product;
		   d_new_label[offset] = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
		
   }

   //check bottom neighbor
   if (y != (yDim - 1)){
		
		neigh = bottom;
		
	   /*begin eval region*/
	   gfunc = 1.0f - ( sqrt ( 

	   					pow( (d_channelOne[offset] - d_channelOne[neigh]), two ) 
	   											   +
	   	                pow( (d_channelTwo[offset] - d_channelTwo[neigh]), two )
	   	                						   +
	   	                pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
	   	                						   +
	   	                pow( (d_channelFour[offset] - d_channelFour[neigh]), two )
	   					
	   					)

	               / max_I);

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered 
	   if( ( eval_product > d_strength[offset])){
	   
		   d_new_strength[offset] = eval_product;
		   d_new_label[offset] = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
   }

   //check front neighbor
   if(z != 0){
		
		neigh = front;

	   /*begin eval region*/
	   gfunc = 1.0f - ( sqrt ( 

	   					pow( (d_channelOne[offset] - d_channelOne[neigh]), two ) 
	   											   +
	   	                pow( (d_channelTwo[offset] - d_channelTwo[neigh]), two )
	   	                						   +
	   	                pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
	   	                						   +
	   	                pow( (d_channelFour[offset] - d_channelFour[neigh]), two )
	   					
	   					)

	               / max_I);

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered 
	   if( ( eval_product > d_strength[offset])){
	   
		   d_new_strength[offset] = eval_product;
		   d_new_label[offset] = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
		
   }

   //check back neighbor
   if(z != (zDim - 1)){
		
		neigh=back;

	   /*begin eval region*/
	   gfunc = 1.0f - ( sqrt ( 

	   					pow( (d_channelOne[offset] - d_channelOne[neigh]), two ) 
	   											   +
	   	                pow( (d_channelTwo[offset] - d_channelTwo[neigh]), two )
	   	                						   +
	   	                pow( (d_channelThree[offset] - d_channelThree[neigh]), two )
	   	                						   +
	   	                pow( (d_channelFour[offset] - d_channelFour[neigh]), two )
	   					
	   					)

	               / max_I);

	   eval_product = gfunc * d_strength[neigh];

	   //if the current cell is overpowered 
	   if( ( eval_product > d_strength[offset])){
	   
		   d_new_strength[offset] = eval_product;
		   d_new_label[offset] = d_label[neigh];
	       
	       d_still_updating[offset] = 1;
       
	   }
	   /*end eval region*/
		
	}
}


	
